#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <algorithm>
using namespace std;

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <sys/time.h>
#include <unistd.h>

// time stamp function in seconds 
double getTimeStamp() {     
    struct timeval  tv ;     
    gettimeofday( &tv, NULL ) ;    
    return (double) tv.tv_usec/1000000 + tv.tv_sec ; 
} 


#define NUM_THREADS 1024

int features = 1024;
int sampels = 10000;
int classes = 10;

float ** training_x1; //3500 * 784
float ** training_x; //3500 * 784
float ** training_y; //3500 * 1
float ** testing_x; //145 * 784
float ** testing_y; //145 * 1


float ** label_onehot; //3500 * 10

void getData(float * res, char buff[])
{
    char *token = strtok(buff," ,");
    int counter=0;
    
    while( token != NULL )
    {
        counter++;
        res[counter-1] = atof(token);
        token = strtok(NULL," ,");
    }
}

void readCSV(char* file, float** mat, int x_dim, int y_dim)
{
    FILE* stream = fopen(file, "r");
    int size_per_pic = y_dim * 30;
    char line[size_per_pic];
    int num;
    if (stream == NULL) {
        perror ("Error opening file");
        return;
    }

    int i = 0;
    while (fgets(line, size_per_pic, stream))
    {
        char* tmp = strdup(line);
        getData(mat[i], tmp);
        i++;
    }
}

void malloc_host(void){
    training_x1 = (float**)malloc(sizeof(float*) * 10000);
    for(int i = 0; i < 10000; i++){
        training_x1[i] = (float*)malloc(sizeof(float) * 1024);
    }

    training_x = (float**)malloc(sizeof(float*) * 10000);
    for(int i = 0; i < 10000; i++){
        training_x[i] = (float*)malloc(sizeof(float) * 1024);
    }

    training_y = (float**)malloc(sizeof(float*) * 10000);
    for(int i = 0; i < 10000; i++){
        training_y[i] = (float*)malloc(sizeof(float) * 1);
    }

    testing_x = (float **)malloc(sizeof(float*) * 2000);
    for(int i = 0; i < 2000; i++){
        testing_x[i] = (float*)malloc(sizeof(float) * 1024);
    }

    testing_y = (float **)malloc(sizeof(float*) * 2000);
    for(int i = 0; i < 2000; i++){
        testing_y[i] = (float*)malloc(sizeof(float) * 1);
    }

    label_onehot = (float **)malloc(sizeof(float*) * 10000);
    for (int i = 0; i < 10000; i++)
    {
        label_onehot[i] = (float*)malloc(sizeof(float) * 10);
    }
}

__global__ void Mult_GPU( float *a,  float *b, float *result,  const int M, const int N, const int S) // M should be batch size
{
    int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < M * S)
    {
        int row = threadId / S;
        int column = threadId % S;
        float temp=0;//reduce global mem access number
        result[threadId] = 0;
        for (int i = 0; i < N; i++)
        {
            //result[threadId] += a[row * N + i] * b[i * S + column];
            temp += a[row * N + i] * b[i * S + column];
        }
        result[threadId]=temp;
    }
}

__global__ void softmax_sum( float *predict, float *sum, const int label_size, const int data_size ){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data_size){
        float temp=0;
        for(int i = 0; i < label_size; i++){
            temp += exp(predict[tid * label_size + i]);
        }
        sum[tid]=temp;
    }
}

__global__ void max( float *predict, float *max, const int label_size, const int data_size ){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < data_size){
        for(int i = 0; i < label_size; i++){
            int max_index = 0;
            max[tid] =  predict[tid * label_size];
            if(predict[tid * label_size + max_index] <  predict[tid * label_size + i]){
                max[tid] = predict[tid * label_size + i];
            }
        }
    }
}

__global__ void normalize(float *predict, float *max, const int label_size, const int data_size){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data_size){
        for(int i = 0; i < label_size; i++){
            predict[tid * label_size + i] -= max[tid];
        }
    }
}

__global__ void softmax( float *softmax_value, float *predict, float *sum,const int label_size, const int data_size ){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data_size){
        for(int i = 0; i < label_size; i++){
            softmax_value[tid * label_size + i] =  exp(predict[tid * label_size + i]) / sum[tid];
        }
    }
}

__global__ void dz(float *softmax_value, float *label, float *dz, const int label_size, const int data_size){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data_size){
        
        for(int i = 0; i < label_size; i++){
            dz[tid * label_size + i] = softmax_value[tid * label_size + i] - label[tid * label_size + i];
        }
    }
}

__global__ void grad(float *train_data, float *dz, float *grad, const int label_size, const int data_size, const int weight_size){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < weight_size){
        for(int i = 0; i < label_size; i++){
            float temp = grad[tid * label_size + i];
            for(int j = 0; j < data_size; j++){
                // grad[tid * label_size + i] += train_data[j * weight_size + tid] * dz[j * label_size + i];
                temp += train_data[j * weight_size + tid] * dz[j * label_size + i];
            }
            grad[tid * label_size + i] = temp;
        }
    }
}

__global__ void weight_update(float *weight, float *grad, const int label_size, const int weight_size, const float learning_rate){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < weight_size){
        for(int i = 0; i < label_size; i++){
            grad[tid * label_size + i] /= 200;
            weight[tid * label_size + i] -= (learning_rate * grad[tid * label_size + i]);
        }
    }
}

__global__ void initialize_dz(float *dz, const int label_size, const int data_size){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data_size){
        for(int i = 0; i < label_size; i++){
            dz[tid * label_size + i] = 0;
        }
    }
}

__global__ void initialize_grad(float *grad, const int label_size, const int weight_size){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < weight_size){
        for(int i = 0; i < label_size; i++){
            grad[tid * label_size + i] = 0;
        }
    }
}

__global__ void initialize(float *sum, float *predict, const int data_size, const int label_size){
    int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data_size){
        sum[tid] = 0;
        for(int i = 0; i < label_size; i++){
            predict[tid * label_size + i] = 0;
        }
    }

}

int randint(int l,int u)
{
  int temp;
  srand((unsigned)time(NULL));
  temp = floor(l + (1.0*rand()/RAND_MAX)*(u - l + 1 ));
  return temp;
}


void random_shuffle(float *data, float *label){
    int len = 10000;
    for (int i = 0 ; i < len; i++) {
        int rand = randint(i, len - 1);
        // swap
        for(int j = 0; j < 1024; j++){
            //swap(data[i][j], arr[rand][j]);
            swap(data[i * 1024 + j], data[rand * 1024 + j]);
        }
        for(int k = 0; k < 10; k++){
            //swap(data[i][j], arr[rand][j]);
            swap(label[i * 10 + k], label[rand * 10 + k]);
        }
    }
}

void data_transpose(float *data1, float *data2){
    int batch_size = 200;
    int weight_size = 1024;
    int label_size = 10;
    for(int i = 0; i < batch_size; i++){
        for(int j = 0; j < weight_size; j++){
            data2[j * batch_size+ i] = data1[i * weight_size + j];
        }
    }
}

void devide_data(float *data1, float *data2, float *label1, float *label2, int index){
    int batch_size = 200;
    int weight_size = 1024;
    int label_size = 10;
    for(int i = 0; i < batch_size; i++){
        for(int j = 0; j < weight_size; j++){
            data1[i * weight_size + j] = data2[(index * batch_size + i) * weight_size + j];
        }
    }
    for(int i = 0; i < batch_size; i++){
        for(int j = 0; j < label_size; j++){
            label1[i * label_size + j] = label2[(index * batch_size  + i) * label_size + j];
        }
    }
}


int main(){
    // Stream
    hipDeviceProp_t prop;
	int deviceID;
	hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    
	if (!prop.deviceOverlap)
	{
		printf("No device will handle overlaps. so no speed up from stream.\n");
		return 0;
    }



    // malloc_host();
    malloc_host();
    readCSV("training_x.csv", training_x, 10000,1024);
    readCSV("training_y.csv", training_y, 1024, 1);
    readCSV("testing_x.csv", testing_x, 2000, 1024);
    readCSV("testing_y.csv", testing_y, 2000, 1);

    readCSV("training_x.csv", training_x1, 10000,1024);

    float learning_rate = 0.1;
    int iter = 1;
    int batch_size = 200;
    int epochs = 50;

    int data_size = 10000;
    int label_size = 10;
    int weight_size = 1024;

    int train_data_bytes = 10000 * 1024 * sizeof(float);
    int batch_data_bytes = 200 * 1024 * sizeof(float);
    int weight_bytes = 1024 * 10 * sizeof(float);
    int predict_bytes = 10000 * 10 * sizeof(float);
    int batch_predict_bytes = 200 * 10 * sizeof(float);
    

    float *h_train_data = (float *) malloc( train_data_bytes ) ;
    float *h_train_data_T = (float *) malloc( train_data_bytes ) ;
    float *h_batch_data = (float *) malloc( batch_data_bytes ) ;
    float *h_batch_data_T = (float *) malloc( batch_data_bytes ) ;
    float *h_label_onehot = (float *) malloc( predict_bytes ) ;
    float *h_batch_label = (float *) malloc( batch_predict_bytes ) ;

    float *h_weight = (float *) malloc( weight_bytes ) ;
    float *h_predict = (float *) malloc( batch_predict_bytes ) ;
    float *h_max = (float *) malloc( 200 * sizeof(float) ) ;
    float *h_sum = (float *) malloc( 200 * sizeof(float) ) ;
    float *h_softmax = (float *) malloc( batch_predict_bytes ) ;
    float *h_dz = (float *) malloc( batch_predict_bytes ) ;
    float *h_grad = (float *) malloc( weight_bytes ) ;


    ////////////////////// Initialize //////////////////////
    ////////////////////// One Hot //////////////////////
    for(int i = 0; i < data_size; i++){
        for(int j = 0; j < weight_size; j++){
            h_train_data_T[j * 10000 + i] = training_x[i][j];
        }
    }

    for(int i = 0; i < data_size; i++){
        label_onehot[i][(int(training_y[i][0] - 1))] = 1;
    }

    for(int i = 0; i < data_size; i++){
        for(int j = 0; j < label_size; j++){
            h_label_onehot[i * label_size + j] = label_onehot[i][j];
        }
    }

    for(int i = 0; i < data_size; i++){
        for(int j = 0; j < weight_size; j++){
            h_train_data[i * weight_size + j] = training_x[i][j];
        }
    }

    for(int i = 0; i < weight_size; i++){
        for(int j = 0; j < label_size; j++){
            h_weight[i * label_size + j] = 1 ;
        }
    }
    //////////////////// Initialize //////////////////////


    ///////////////////////////////// GPU_SIDE ///////////////////////////////////
    float *d_train_data, *d_train_data_T, *d_label, * d_weight, *d_predict, *d_predict_sum, *d_sum, *d_max, *d_softmax_value;
    float *d_dz, *d_grad;
    float *d_batch_data, *d_batch_data_T, *d_batch_label;

    hipGetErrorString(hipMalloc( (void **) &d_train_data, train_data_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_train_data_T, batch_data_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_batch_data, batch_data_bytes )) ;
    hipGetErrorString(hipMalloc( (void **) &d_batch_data_T, train_data_bytes )) ;
    
    hipGetErrorString(hipMalloc( (void **) &d_batch_label, batch_predict_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_predict, batch_predict_bytes)) ;

    hipGetErrorString(hipMalloc( (void **) &d_weight, weight_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_sum, 200 * sizeof(float))) ;
    hipGetErrorString(hipMalloc( (void **) &d_softmax_value, batch_predict_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_dz, batch_predict_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_grad, weight_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_max, 200 * sizeof(float))) ;

    // //Configure blockDim
    int bdx = 32, bdy = 32;
    while(data_size > bdx * 65535)
    {
        bdx = bdx * 2;
        bdy = bdy / 2;
    }
    while(weight_size > bdy * 65535)
    {
        bdy = bdy * 2;
        bdx = bdx / 2;
    }
    dim3 blockDim( bdx,bdy ) ; // you will want to configure this
    dim3 gridDim( (int)((data_size + blockDim.x-1)/blockDim.x), (int)((weight_size + blockDim.y-1)/blockDim.y) ) ;

    //////////////////////////////// invoke Kernel (Logistic Regression) ////////////////////////////////
    double timeStamp1 = getTimeStamp() ;

    for(int train  = 0; train < 1000; train++){
        //////////////////////Random shuffle data/////////////////////////////
        random_shuffle(h_train_data, h_label_onehot);

        for(int epoch = 0; epoch < epochs; epoch++){
            //////////////////////   Transfer data   ////////////////////////////
            devide_data(h_batch_data, h_train_data, h_batch_label, h_label_onehot, epoch);
            hipGetErrorString(hipMemcpyAsync( d_weight, h_weight, weight_bytes, hipMemcpyHostToDevice)) ;
            hipGetErrorString(hipMemcpyAsync( d_batch_data, h_batch_data, batch_data_bytes, hipMemcpyHostToDevice)) ;
            hipGetErrorString(hipMemcpyAsync( d_batch_label, h_batch_label, batch_predict_bytes, hipMemcpyHostToDevice)) ;
            

            //////////////////////     Computation    ///////////////////////////
            //Initialize
            initialize<<<gridDim, blockDim, 0>>>(d_sum, d_predict, batch_size, label_size);
            hipGetErrorString(hipDeviceSynchronize());
            initialize_dz<<<gridDim, blockDim, 0>>>(d_dz, label_size, batch_size);
            hipGetErrorString(hipDeviceSynchronize());
            initialize_grad<<<gridDim, blockDim, 0>>>(d_grad, label_size, weight_size);
            hipGetErrorString(hipDeviceSynchronize());

            //DOT
            Mult_GPU<<<gridDim, blockDim, 0>>>( d_batch_data, d_weight, d_predict, batch_size, weight_size, label_size) ;
            hipGetErrorString(hipDeviceSynchronize());

            max<<<gridDim, blockDim, 0>>>( d_predict, d_max, label_size, batch_size );
            hipGetErrorString(hipDeviceSynchronize());	

            normalize<<<gridDim, blockDim, 0>>>(d_predict, d_max, label_size, batch_size);
            hipGetErrorString(hipDeviceSynchronize());

            // Softmax
            softmax_sum<<<gridDim, blockDim, 0>>>( d_predict, d_sum, label_size, batch_size );
            hipGetErrorString(hipDeviceSynchronize());
            softmax<<<gridDim, blockDim, 0>>>( d_softmax_value, d_predict, d_sum, label_size, batch_size );
            hipGetErrorString(hipDeviceSynchronize());

            // Weight Update
            dz<<<gridDim, blockDim, 0>>>(d_softmax_value, d_batch_label, d_dz, label_size, batch_size);
            hipGetErrorString(hipDeviceSynchronize());
            grad<<<gridDim, blockDim, 0>>>(d_batch_data, d_dz, d_grad, label_size, batch_size, weight_size);
            hipGetErrorString(hipDeviceSynchronize());
            weight_update<<<gridDim, blockDim, 0>>>(d_weight, d_grad, label_size, weight_size, learning_rate);
            hipGetErrorString(hipDeviceSynchronize());

       }
   }
   double timeStamp2 = getTimeStamp() ;
   

    // ///////////////////////////// Test /////////////////////////////////////
    // hipGetErrorString(hipMemcpyAsync( h_predict, d_predict, batch_predict_bytes, hipMemcpyDeviceToHost, stream )) ;

    hipGetErrorString(hipMemcpyAsync( h_weight, d_weight, weight_bytes, hipMemcpyDeviceToHost)) ;

    for(int i = 0; i < weight_size; i++){
        for(int j = 0; j < label_size; j++){
            printf("h_weight: %f\n", h_weight[i * label_size + j]);
        }
    }

    printf("%.6f\n", timeStamp2-timeStamp1);

    // Test case
    // for(int i = 0; i < data_size; i++){
    //     for(int j = 0; j < weight_size; j++){
    //         h_train_data[i * weight_size + j] = training_x1[i][j];
    //         //printf(" h_train_data: %f\n",  h_train_data[i * label_size + j]);
    //     }
    // }

    float *h_test_predict = (float *) malloc( predict_bytes ) ;
    float *h_test_max= (float *) malloc( 10000 * sizeof(float) ) ;
    float *h_test_sum= (float *) malloc( 10000 * sizeof(float) ) ;

    float *d_test_predict, *d_test_max, *d_test_sum, *d_test_softmax;
    hipGetErrorString(hipMalloc( (void **) &d_test_predict, predict_bytes)) ;
    hipGetErrorString(hipMalloc( (void **) &d_test_sum, 10000 * sizeof(float))) ;
    hipGetErrorString(hipMalloc( (void **) &d_test_max, 10000 * sizeof(float))) ;
    hipGetErrorString(hipMalloc( (void **) &d_test_softmax, predict_bytes)) ;


    hipGetErrorString(hipMemcpy( d_train_data, h_train_data, train_data_bytes, hipMemcpyHostToDevice )) ;
    hipGetErrorString(hipMemcpy( d_weight, h_weight, weight_bytes, hipMemcpyHostToDevice )) ;

    Mult_GPU<<<gridDim, blockDim>>>( d_train_data, d_weight, d_test_predict, data_size, weight_size, label_size) ;
    hipGetErrorString(hipDeviceSynchronize());
    max<<<gridDim, blockDim>>>( d_test_predict, d_test_max, label_size, data_size );
    hipGetErrorString(hipDeviceSynchronize());	
    normalize<<<gridDim, blockDim>>>(d_test_predict, d_test_max, label_size, data_size);
    hipGetErrorString(hipDeviceSynchronize());

    softmax_sum<<<gridDim, blockDim, 0>>>( d_test_predict, d_test_sum, label_size, data_size );
    hipGetErrorString(hipDeviceSynchronize());
    softmax<<<gridDim, blockDim, 0>>>( d_test_softmax, d_test_predict, d_test_sum, label_size, data_size );
    hipGetErrorString(hipDeviceSynchronize());

    hipGetErrorString(hipMemcpy(h_test_predict, d_test_softmax, predict_bytes, hipMemcpyDeviceToHost )) ;

    // float total_error = 0;
    // for(int i = 0; i < data_size; i++){
    //     for(int j = 0; j < label_size; j++){
    //         total_error += (h_label_onehot[i * label_size + j] * h_test_predict[i * label_size + j]);
    //     }
    // }
    // printf("error: %f\n", total_error );

    // hipGetErrorString(hipMemcpy(h_test_sum, d_test_sum, 10000 * sizeof(float), hipMemcpyDeviceToHost )) ;
    // for(int i = 0; i < 10000; i++){
    //     printf("h_max: %f\n", h_test_sum[i]);
    // }

    // hipGetErrorString(hipMemcpy(h_test_predict, d_test_softmax, predict_bytes, hipMemcpyDeviceToHost )) ;

    // for(int i = 0; i < 10000; i++){
    //     for(int j = 0; j < 10; j++){
    //         printf("h_predict: %f\n", h_test_predict[i * label_size + j]);
    //     }
    // }



    ///////////////////////// Error ///////////////////////////////
    // float total_error = 0;
    // for(int i = 0; i < batch_size; i++){
    //     for(int j = 0; j < label_size; j++){
    //         total_error -= label_onehot[i][j] * log(h_softmax[i * label_size + j]) ;
    //     }
    // }
    // printf("error: %f\n", total_error );

}